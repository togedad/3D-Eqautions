
#include <hip/hip_runtime.h>
//#include <stdio.h>
//#include <iostream>
//#include <iomanip>
//
//#include "SDL.h"
//#include <cuda_runtime.h>
//#include "cuda_kernel.cuh"
//#include "device_launch_parameters.h"
////#include "equation.cuh"
//
////other
//SDL_Surface* surf;
//int defaultSize = 75;
//
////char* temp = new char[len] { 'T', '(', 'S', '(', 'y', ')', ')'};
//char* GPUequations;
//
//int* equaitonLens;
//int* valusesInEquationLens;
//int* hits;
//
//float* vals;
//float* valusesInEquation;
//
//float resalution;
//
//int totalPoints;
//int blocks;
//
//int screenHeight;
//int screenWidth;
//int renderDistance;
//
//#pragma region Consts
//
//
//const int threadsBeingUsed = 256;
//
//const int operationsLen = 8;
//const int oneInputOperationsLen = 3;
//const int towInputOperationsLen = 4;
//
//const int numberToAddOnForAskiiRefrance = 91; // this will need to cahnge eventually
//
//__device__
//char operations[operationsLen] = { '*', '/', '+', '-', 'S', 'C', 'T', '(' };
//__device__
//char oneInputOperations[oneInputOperationsLen] = { 'S', 'C', 'T' };
//__device__
//char towInputOperations[towInputOperationsLen] = { '*', '/', '+', '-' };
////device funtions
//
//#pragma endregion
//
//
////to draw pixles to a surface
//void drawPixel(SDL_Surface* surface, int x, int y, uint8_t R, uint8_t G, uint8_t B) {
//    SDL_LockSurface(surface);
//    uint8_t* pixelArr = (uint8_t*)surface->pixels;
//    pixelArr[y * surface->pitch + (x * surface->format->BytesPerPixel)] = B;
//    pixelArr[y * surface->pitch + (x * surface->format->BytesPerPixel) + 1] = G;
//    pixelArr[y * surface->pitch + (x * surface->format->BytesPerPixel) + 2] = R;
//
//    SDL_UnlockSurface(surface);
//    //remmeber you need to update windop surface after use 
//
//}
//
////these are the same as in char functions but they have been re wirrten for use in the gpu
////
////
////if (threadIdx.x == 0 && blockIdx.x == 0) {
////    printf("\n out of sin");
////    outputEquation(equationPart, equationPartLen);
////    printf("\n");
////
////}
//
//void CPUoutputEquation(char* string, int len) { //looops through ll charicters and prints tehm to screan 
//    for (int i = 0; i < len; ++i) {
//        char temp = string[i];
//        printf("%c", temp);
//    }
//}
//
//#pragma region Equation
//
//__device__
//void outputEquation(char* string, int len) { //looops through ll charicters and prints tehm to screan 
//    for (int i = 0; i < len; ++i) {
//        char temp = string[i];
//        printf("%c", temp);
//    }
//}
//
//__device__
//void remove(char*& string, int& len, int start, int end) {
//    int removeLengh = end - start;
//    int newLen = len - removeLengh;//finds the new lengh of arrray
//
//    char* newCharArr = string;//copying the array
//
//    for (int i = end; i < len; i++) {
//
//        string[i - removeLengh] = newCharArr[i];//adding charicter to the list
//    }
//
//    len = newLen;//changeing len of the string to correct value
//}
//
//__device__
//void reverse(char*& string, int& len) {
//    char* copyCharArr = new char[len];
//
//    for (int i = 0; i < len; i++) {
//        char letter = string[len - 1 - i];
//        copyCharArr[i] = string[len - 1 - i];//taking alst element pos and taking i to reverse the array
//    }
//
//    string = copyCharArr;
//}
//
//__device__
//void insert(char*& string, int& len, char* insertStr, int insertLen, int location) {
//    int newLen = len + insertLen;//finds the new lengh of arrray
//    char* newCharArr = new char[len];
//
//    if (location == len) // end of string
//    {
//        newCharArr = string;
//
//        for (int n = 0; n < insertLen; n++) {//inserting new string 
//            newCharArr[len + n] = insertStr[n];
//        }
//    }
//    else {
//        int add = 0;//add holds the number of time you have to skip so when you get to the charicters that have been inserted so the dont get written over straight away
//
//        for (int i = 0; i < len; i++) {
//
//            if (i == location) {//looking when to insert
//                add = insertLen;
//                for (int n = 0; n < insertLen; n++) {//inserting new string 
//                    newCharArr[i + n] = insertStr[n];
//                }
//            }
//
//            newCharArr[i + add] = string[i];
//        }
//    }
//
//    string = newCharArr;
//    len = newLen;//setting new lengh
//
//}
//
//__device__
//void pushToBack(char*& string, int& len, char charToPush) {
//    len++;//finds the new lengh of arrray
//    string[len - 1] = charToPush;//adding flaots form insert string to string   
//}
//
//__device__
//void pushToBackFloat(float*& numbers, int& len, float floatToPush) {
//    len++;//finds the new lengh of arrray
//    numbers[len - 1] = floatToPush;//adding flaots form insert string to string   
//}
//
//__device__
//void replace(char*& string, int& len, char* removeStr, int removeLen, char* replacetStr, int replacetLen) {
//    for (int i = 0; i < len; i++) {
//        if (string[i] == removeStr[0]) {//looking to see if charicter is same as first charicter of string to be replaced
//            bool found = true;
//
//            for (int n = 1; n < removeLen; n++) {//checking too see if it is the sub string that is to be removed
//                if (string[i + n] != removeStr[n]) {//sinmple loop chaeck too see if it is the correct sub string
//                    found = false;
//                    break;
//                }
//            }
//
//            if (found) {
//                remove(string, len, i, i + removeLen);//removes string
//                insert(string, len, replacetStr, replacetLen, i);//inserts new string 
//                i += replacetLen;//skips i over the new string, as in some cases new string could include replacement string so could create infinate loop
//            }
//        }
//    }
//}
//
//__device__
//float returnEquationResult(char* equationPart, int equationPartLen, float* numbersInEquation, int& numbersLen) { //some funciton will be slow and i think are being called multiple times as it runns really slow when it is just renterng y
//
//    #pragma region one val ops
//
//    int depth = 0;
//    char* tempStr =new char[100];
//    int tempStrLen = 0;
//
//    for (int i = 0; i < equationPartLen; i++) {
//        char letter = equationPart[i];
//
//        if (letter == '(') {//seeing if it is in a set of brackets - the depth is howmany sets of brackes you are in for example ( (v) ) teh v is in a depth of 2 as it is 2 layers deep we only want to compute numbers on lowest level (0) as the numbers on loweelevels are what we are looking at teh brakets in highleves are fed in the prgram with recution 
//            depth++;
//        }
//        else if (letter == ')') {//seeing if it is leaveing a set of brackets
//            depth--;
//        }
//        else {
//            if ((depth == 0)) { //only runs if it is on right level look at comments above for depth in more detail
//
//                int start = i;
//
//                //outputEquation(equationPart, equationPartLen);
//
//                for (int n = 0; n < oneInputOperationsLen; n++) {//divided by size of char as it is not the amount of elemets but the size of all the data (types) of elemets so dividing it by the data type makes it the right size
//                    
//                    if (letter == oneInputOperations[n]) {
//                        
//
//                        i += 2;
//                        depth++; //as you skip over a a open bracket that needs to be acounted for
//
//                        while (depth > 0) {//needs an exit condition if no bracket -------------------------------------------------------------------------------------------!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
//                            
//                            if (equationPart[i] == '(') {
//                                depth++;
//                            }
//                            else if (equationPart[i] == ')') {
//                                depth--;
//                            }
//
//                            if (depth > 0) {
//                                char tempLetter = equationPart[i];
//                                pushToBack(tempStr, tempStrLen, tempLetter);
//                            }
//
//                            i++;//referts to the for loop i if it seems out of place 
//                        }
//
//                        remove(equationPart, equationPartLen, start, i);
//
//                        double val;//declaring now so is=t can be useing in any of the cases below - this is BAD CODE as it uses 3 switch cases, if more opreations want to be added then a  better system should be devised but for now this is all that is needed
//                        val = returnEquationResult(tempStr, tempStrLen, numbersInEquation, numbersLen);
//
//                        switch (letter) {
//                        case 'S':
//                            val = sin(val);
//                            break;
//                        case 'C':
//                            val = cos(val);
//                            break;
//                        case 'T':
//                            val = tan(val);
//                            break;
//                        }
//
//                        char* temp = new char[2];
//                        temp[0] = '~';
//                        temp[1] = numbersLen + numberToAddOnForAskiiRefrance;//the value of the askicharicter is the index of nubers to look at
//
//                        pushToBackFloat(numbersInEquation, numbersLen, val);//push the float val to the back of this 
//                        insert(equationPart, equationPartLen, temp, 2, start);
//
//                        i = start + 2;
//                    }
//                }
//            }
//        }
//
//        //looking for times and multiplication
//    }
//
//    #pragma endregion
//
//    #pragma region mul divs
//
//    //looking at times and divide operations
//    depth = 0;
//    tempStr = new char[100];
//    tempStrLen = 0;
//
//    for (int i = 0; i < equationPartLen; i++) {
//        char letter = equationPart[i];
//
//        if (letter == '(') {//seeing if it is in a set of brackets - the depth is howmany sets of brackes you are in for example ( (v) ) teh v is in a depth of 2 as it is 2 layers deep we only want to compute numbers on lowest level (0) as the numbers on loweelevels are what we are looking at teh brakets in highleves are fed in the prgram with recution 
//            depth++;
//        }
//        else if (letter == ')') {//seeing if it is leaveing a set of brackets
//            depth--;
//        }
//
//        if (depth == 0 && (letter == '*' || letter == '/')) {
//
//
//            int currentIndex = i;
//
//            double value1;//value on right of symbol
//            double value2;//value on left of symbol
//
//            // for removeing the necerry information
//            int end;
//            int start;
//
//            //looking infront of the symbol
//            i++;
//            if (equationPart[i] == '(') {
//                int currentIndex = i;
//                i++;
//                depth++; //as you skip over a a open bracket that needs to be acounted for
//
//                while (depth > 0) {
//                    if (equationPart[i] == '(') {
//                        depth++;
//                    }
//                    else if (equationPart[i] == ')') {
//                        depth--;
//                    }
//
//                    if (depth > 0) {
//                        pushToBack(tempStr, tempStrLen, equationPart[i]);
//                    }
//                    i++;//referts to teh for loop i if it seems out of place 
//                }
//
//                value1 = returnEquationResult(tempStr, tempStrLen, numbersInEquation, numbersLen);
//                tempStrLen = 0;
//                end = i;
//                i = currentIndex;
//                //break;
//            }
//            else
//            {
//                end = i + 2;
//
//                char* stringVal = new char[2];
//                stringVal[0] = equationPart[i];
//                stringVal[1] = equationPart[i + 1];
//                value1 = returnEquationResult(stringVal, 2, numbersInEquation, numbersLen);
//
//                //printf("%i \n", stringVal[1] - numberToAddOnForAskiiRefrance);
//            }
//
//            //looking behind of the symbol
//            i--;
//            if (equationPart[i] == ')') {
//                i--;
//                depth--; //as you skip over a a open bracket that needs to be acounted for
//
//                while (depth < 0) {
//                    if (equationPart[i] == '(') {
//                        depth++;
//                    }
//                    else if (equationPart[i] == ')') {
//                        depth--;
//                    }
//
//                    if (depth < 0) {
//                        pushToBack(tempStr, tempStrLen, equationPart[i]);
//                    }
//                    i--;//referts to teh for loop i if it seems out of place 
//                }
//
//                reverse(tempStr, tempStrLen);
//
//                value2 = returnEquationResult(tempStr, tempStrLen, numbersInEquation, numbersLen);
//
//                tempStrLen = 0;
//                start = i;
//                //break;
//            }
//            else
//            {
//                start = i - 2;
//
//                char* stringVal = new char[2];
//                stringVal[0] = equationPart[i - 2];
//                stringVal[1] = equationPart[i - 1];
//
//                value2 = returnEquationResult(stringVal, 2, numbersInEquation, numbersLen);
//
//
//            }
//                        
//            remove(equationPart, equationPartLen, start, end);
//
//            char* temp = new char[2];
//            temp[0] = '~';
//            temp[1] = numbersLen + numberToAddOnForAskiiRefrance;//the value of the askicharicter is the index of nubers to look at
//
//            //printf("%i %g %g | %g %g | %i \n", temp[1] - numberToAddOnForAskiiRefrance, value2, value1, numbersInEquation[4], numbersInEquation[2], &numbersInEquation[4]);//value2, value1, equationPart[1] - numberToAddOnForAskiiRefrance, &numbersInEquation[4], temp[1] - numberToAddOnForAskiiRefrance, 'A');
//
//            if (letter == '*') {
//                value2 = (float)value2 * value1;
//            }
//            else if (letter == '/') {
//                value2 = (float)value2 / value1;
//            }
//
//            i = start;
//
//            pushToBackFloat(numbersInEquation, numbersLen, value2);//push the float val to the back of this 
//            insert(equationPart, equationPartLen, temp, 2, start);
//
//
//            tempStrLen = 0;
//        }
//    }
//
//    #pragma endregion
//
//    #pragma region add sub
//
//    //looking at subtract and add operations
//    depth = 0;
//    tempStr = new char[100];
//    tempStrLen = 0;
//
//    for (int i = 0; i < equationPartLen; i++) {
//        char letter = equationPart[i];
//
//        if (letter == '(') {//seeing if it is in a set of brackets - the depth is howmany sets of brackes you are in for example ( (v) ) teh v is in a depth of 2 as it is 2 layers deep we only want to compute numbers on lowest level (0) as the numbers on loweelevels are what we are looking at teh brakets in highleves are fed in the prgram with recution 
//            depth++;
//        }
//        else if (letter == ')') {//seeing if it is leaveing a set of brackets
//            depth--;
//        }
//
//        if (depth == 0 && (letter == '+' || letter == '-')) {
//
//            int currentIndex = i;
//
//            double value1;//value on right of symbol
//            double value2;//value on left of symbol
//
//            // for removeing the necerry information
//            int end;
//            int start;
//
//            //looking infront of the symbol
//            i++;
//            if (equationPart[i] == '(') {
//                int currentIndex = i;
//                i++;
//                depth++; //as you skip over a a open bracket that needs to be acounted for
//
//                while (depth > 0) {
//                    if (equationPart[i] == '(') {
//                        depth++;
//                    }
//                    else if (equationPart[i] == ')') {
//                        depth--;
//                    }
//
//                    if (depth > 0) {
//                        pushToBack(tempStr, tempStrLen, equationPart[i]);
//                    }
//                    i++;//referts to teh for loop i if it seems out of place 
//                }
//
//                value1 = returnEquationResult(tempStr, tempStrLen, numbersInEquation, numbersLen);
//                tempStrLen = 0;
//                end = i;
//                i = currentIndex;
//                //break;
//            }
//            else
//            {
//                end = i + 2;
//
//                char* stringVal = new char[2];
//                stringVal[0] = equationPart[i];
//                stringVal[1] = equationPart[i + 1];
//                value1 = returnEquationResult(stringVal, 2, numbersInEquation, numbersLen);
//            }
//
//            //looking behind of the symbol
//            i--;
//            if (equationPart[i] == ')') {
//                i--;
//                depth--; //as you skip over a a open bracket that needs to be acounted for
//
//                while (depth < 0) {
//                    if (equationPart[i] == '(') {
//                        depth++;
//                    }
//                    else if (equationPart[i] == ')') {
//                        depth--;
//                    }
//
//                    if (depth < 0) {
//                        pushToBack(tempStr, tempStrLen, equationPart[i]);
//                    }
//                    i--;//referts to teh for loop i if it seems out of place 
//                }
//
//                reverse(tempStr, tempStrLen);
//
//                value2 = returnEquationResult(tempStr, tempStrLen, numbersInEquation, numbersLen);
//
//                tempStrLen = 0;
//                start = i;
//                //break;
//            }
//            else
//            {
//                start = i - 2;
//
//                char* stringVal = new char[2];
//                stringVal[0] = equationPart[i - 2];
//                stringVal[1] = equationPart[i - 1];
//                value2 = returnEquationResult(stringVal, 2, numbersInEquation, numbersLen);
//            }
//
//            remove(equationPart, equationPartLen, start, end);
//
//            
//
//            if (letter == '+') {
//                value2 = (float)value2 + value1;
//            }
//            else if (letter == '-') {
//                value2 = (float)value2 - value1;
//            }
//
//            char* temp = new char[2];
//            temp[0] = '~';
//            temp[1] = numbersLen + numberToAddOnForAskiiRefrance;//the value of the askicharicter is the index of nubers to look at
//
//
//            i = start;
//            pushToBackFloat(numbersInEquation, numbersLen, value2);//push the float val to the back of this 
//            insert(equationPart, equationPartLen, temp, 2, start);
//
//            //#printf("-- %g %i\n", numbersInEquation[4], start);
//
//
//            tempStrLen = 0;
//        }
//    }
//
//    #pragma endregion
//
//    char endLetter = equationPart[0];
//    if (endLetter == '~') {
//        int index = equationPart[1];
//        index -= numberToAddOnForAskiiRefrance;
//
//        return numbersInEquation[index];
//    }
//    else {
//        //printf("failed");
//        return 500;
//    }
//
//
//};
////this should be called to get value of equation not return Eqation Result function as it needs the prper setup
//__device__
//float returnValSetup(char* stringEquation, int& len, float* numbersInEquation, int numbersLen, float x, float y, float z) {//x y z arrays are assumnesd to be lengh of float - 7 significant figures
//
//    float* copyNumberseInEquaiton = new float[numbersLen];
//    int copyNumLen = numbersLen;
//
//    memcpy(copyNumberseInEquaiton, &numbersInEquation[0], numbersLen * sizeof(float));
//
//    char* copyEquaiton = new char[len];
//    int copyEquLen = len;
//
//    memcpy(copyEquaiton, &stringEquation[0], len * sizeof(char));
//
//    copyNumberseInEquaiton[1] = x;//start at 1 as 0 in askii is no letter so you cna never convert it to a letter as there is nothing there 
//    copyNumberseInEquaiton[2] = y;
//    copyNumberseInEquaiton[3] = z;
//
//    return returnEquationResult(copyEquaiton, copyEquLen, copyNumberseInEquaiton, copyNumLen);
//}
//
//#pragma endregion
//
////global functions
//
//__global__
//void test(char* equation, int* len, float* numbersInEquation, int* numbersLen, float* vals, float screenXRotation, float screenYRotation, int screenWidth, int screenHeight, int renderDistance, float zoom, float resalution, int* hitMap)
//{
//    //int totalPoints = screenWidth * screenHeight * (renderDistance / resalution);
//    int gloabalThreadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
//
//    if (gloabalThreadNumber >= screenHeight * screenWidth) { return; }//this is to stop wrong parts of memory being overwritten. e.g. lests say you have 2 bloacks of 10 but only porsesing 15 elemets then there are 5 floats that are going to be written but these floats 
//
//    //NEEDS TO BE CHANGED TO BE HOW FAR ALONG Z AXIS AND THEN GIVIEN ITS ACTUAL VALUE <------------- TO DO
//    //FIND SOLUTION WHERE THESE VALUES ARE ACTUALY CALCULATED SO THAT IT DOESNT HAVE TO BE JSUT A SQUARE
//    float xWidth = renderDistance,
//          yWidth = renderDistance,
//
//    int totalZpoints = renderDistance / resalution;
//
//    int screenX = gloabalThreadNumber % screenWidth,
//        screenY = (gloabalThreadNumber % (screenHeight * screenHeight)) / screenWidth;
//
//     float z = (((gloabalThreadNumber / (screenWidth * screenHeight)) / (float)totalZpoints) * (renderDistance)) - ((renderDistance) / (float)2), // <----------- NOT WORKING ALLWAYS GIVE NEGATIVE A HALF
//           x = (((gloabalThreadNumber % screenWidth) / (float)screenWidth) * (zoom)) - ((zoom) / (float)2),
//           y = ((((gloabalThreadNumber % (screenHeight * screenHeight)) / screenWidth) / (float)screenHeight) * (zoom)) - ((zoom) / (float)2);
//
//    float xT = x * cos(screenXRotation) - z * sin(screenXRotation); //xT means it is the x location that has been transformed inot the corect postion
//    float zT = x * sin(screenXRotation) + z * cos(screenXRotation);
//
//    //ROTAION FOR UP AND DOWN SHALL GO HERE <------ TO DO
//    float yT = y;
//
//    x = xT; //these are so i do not need to create a new varbiles they just hold the lcoation of where the ray starts now.
//    y = yT;
//    z = zT;
//
//    float val = returnValSetup(equation, *len, numbersInEquation, *numbersLen, xT, yT, zT);
//
//
//    //THIS NEEDS TO BE CHANGED TO ONLY RUN ONCE, IF COLITION FOUND THAN CORRECT PLACE IN ARRAY IS INCREMETED BY ONE <------- TO DO
//
//
//    if (val >= -resalution/(float)2 && val <= resalution/(float)2) { //this checks for a colition, if a value is is closengough to zero then it is a coltion, this is determined by checking to see if the abs of the value is withing half of the resalution as that is the distance between points
//       hitMap[gloabalThreadNumber % (screenWidth * screenHeight)]++; //is incremented as i would like to knwo homany hits i get 
//    }
//}
//
//int setUpKernal(Equation equ, int _screenWidth, int _screenHeight, int _renderDistance, float _resalution, SDL_Surface* _surface) {
//    screenWidth = _screenWidth;
//    screenHeight = _screenHeight;
//    surf = _surface;
//    resalution = _resalution;
//    renderDistance = _renderDistance;
//
//    int amountOfImagePixels = screenHeight * screenWidth;
//
//
//    //VALUES NEED TO BE CHANGED TO THE CORRECT AMOUNT OF DATA ALOCATED 
//    cudaDeviceSetLimit(cudaLimitMallocHeapSize, (int)(0.5 * 125000000));
//    //cudaDeviceGetLimit(&size, cudaLimitMallocHeapSize);
//
//
//    cudaDeviceSetLimit(cudaLimitStackSize, 1025);
//    //cudaDeviceGetLimit(&size, cudaLimitStackSize);
//
//
//    //#pragma region alocating memoery for eqautions
//
//    cudaMalloc(&GPUequations, defaultSize * sizeof(char));
//    cudaMalloc(&equaitonLens, defaultSize * sizeof(int));
//
//    cudaMalloc(&valusesInEquation, defaultSize * sizeof(float));
//    cudaMalloc(&valusesInEquationLens, defaultSize * sizeof(int));
//
//    cudaMalloc(&vals, defaultSize * sizeof(float));
//
//    cudaMalloc(&hits, amountOfImagePixels * sizeof(int));
//
//
//    valusesInEquationLens[0] = equ.equationVariblesLen;
//    equaitonLens[0] = equ.equationLen;
//
//    cudaMemcpy(GPUequations, &equ.equation[0], defaultSize * sizeof(char), cudaMemcpyHostToDevice);
//    cudaMemcpy(valusesInEquation, &equ.equationVaribles[0], defaultSize * sizeof(int), cudaMemcpyHostToDevice);
//    cudaMemset(hits, 0, amountOfImagePixels);
//
//    //this calculates the tola points being checked, the rendDist / res is to find how many points to be checked on z axis
//    totalPoints = amountOfImagePixels * (renderDistance / resalution);
//    blocks = totalPoints / threadsBeingUsed;
//
//    return 0;
//
//}
//
//int freeEquaitonMemory() {
//    cudaFree(GPUequations);
//    cudaFree(equaitonLens);
//
//    cudaFree(vals);
//    cudaFree(hits);
//
//    cudaFree(valusesInEquationLens);
//    cudaFree(valusesInEquation);
//
//    return 0;
//}
//
//int renderKernal(float xRoatation, float yRoatation, float zoom)
//{
//
//    //clock_t start, finish;
//    //double totaltime;
//    //start = clock();
//
//    //////the threads being used now needs to include treads on the Z axis <----------------------- DONE I THINK
//
//    ////test <<<blocks, threadsBeingUsed>>>(GPUequations, equaitonLens, valusesInEquation, valusesInEquationLens, vals, xRoatation, yRoatation, screenWidth, screenHeight, renderDistance, zoom, resalution, hits);
//
//    //cudaDeviceSynchronize();
//
//    //int* hitsCopy;
//    //cudaMemcpy(hits, hitsCopy, screenWidth * screenHeight * sizeof(int), cudaMemcpyHostToDevice);
//
//    ////////NEEDS TO BE CHAGED INSTEAD OF VALUE BEING CEHCKED FOR TOLANCE NEEDS TO BE CHECKED TOO SEE IF ITS GREATER THAN 0 <--------------------------- TO DO
//    //for (int y = 0; y < screenHeight; y++) {
//    //    for (int x = 0; x < screenWidth; x++) {
//    //        int val = hitsCopy[(y * screenWidth) + x];
//    //        if (val > 0) {
//    //            drawPixel(surf, x, y, 255, 0, 0);
//    //        }
//    //        //else
//    //        //{
//    //        //    drawPixel(surface, x, y, 255, 255, 255);
//    //        //}
//    //    }
//    //}
//
//
//
//    //finish = clock();
//    //totaltime = (double)(finish - start) / CLOCKS_PER_SEC;
//    //printf("%f", totaltime);
//
//    return 0;
//}
